#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernEfficientScan(int n, int* odata, const int* idata, int* blockSum) {
            int id = blockIdx.x * blockDim.x + threadIdx.x;
            int thid = threadIdx.x;
            int size = 2 * blockDim.x;
            int offset = 1;

            // Shared memory array
            extern __shared__ int temp[];

            // Load input into shared memory
            temp[2 * thid] = idata[2 * id];
            temp[2 * thid + 1] = idata[2 * id + 1];
            __syncthreads();

            // Up sweep
            for (int d = size >> 1; d > 0; d >>= 1) {
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }

            // Set the root for down sweep tree to 0
            if (thid == 0) {
                blockSum[blockIdx.x] = temp[size - 1];
                temp[size - 1] = 0;
            }

            // Down sweep
            for (int d = 1; d < size; d *= 2) {
                offset >>= 1;
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();

            odata[2 * id] = temp[2 * thid]; // write results to device memory
            odata[2 * id + 1] = temp[2 * thid + 1];
        }


        __global__ void kernAddBlockOffset(int n, int* odata, const int* blockOffset) {
            int id = blockIdx.x * blockDim.x + threadIdx.x;
            if (id >= n / 2) return;
            odata[2 * id] += blockOffset[blockIdx.x];
            odata[2 * id + 1] += blockOffset[blockIdx.x];
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            //timer().startGpuTimer();
            // TODO
            // Pad
            int pad_n = 1 << ilog2ceil(n);

            // Assign hreads and blocks
            int threads = 1024;
            int blocks = (pad_n / 2 + threads - 1) / threads; // ceil

            // Allocate & copy memory
            int* d_odata, *d_idata;
            
            hipMalloc(&d_odata, pad_n * sizeof(int));
            checkCUDAError("Efficient::scan::hipMalloc d_odata fails!");

            hipMalloc(&d_idata, pad_n * sizeof(int));
            checkCUDAError("Efficient::scan::hipMalloc d_idata fails!");

            hipMemset(d_idata, 0, pad_n * sizeof(int));
            checkCUDAError("Efficient::scan::hipMemset d_idata fails!");
            
            hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Efficient::scan::hipMemcpyHostToDevice fails!");

            // Handle array of arbitrary length - create and set blockSum
            int* blockSum = new int[blocks];

            int* d_blockSum;
            hipMalloc(&d_blockSum, blocks * sizeof(int));
            checkCUDAError("Efficient::scan::hipMalloc d_blockSum fails!");

            hipMemset(d_blockSum, 0, blocks * sizeof(int)); // Initialize the sum to 0
            checkCUDAError("Efficient::scan::hipMemset d_blockSum fails!");

            // Call kernEfficientScan
            kernEfficientScan<<<blocks, threads, 2 * threads * sizeof(int)>>>(pad_n, d_odata, d_idata, d_blockSum);
            checkCUDAError("Efficient::scan::kernEfficientScan fails!");

            // Handle array of arbitrary length - scan the blockSum (on CPU)
            hipMemcpy(blockSum, d_blockSum, blocks * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Efficient::scan::hipMemcpyDeviceToHost fails!");

            int* blockOffset = new int[blocks];
            blockOffset[0] = 0;
            for (int i = 1; i < blocks; ++i) {
                blockOffset[i] = blockOffset[i - 1] + blockSum[i - 1];
            }

            // Handle array of arbitrary length - add the offset back to array
            int* d_blockOffset;
            hipMalloc(&d_blockOffset, blocks * sizeof(int));
            checkCUDAError("Efficient::scan::hipMalloc d_blockOffset fails!");

            hipMemcpy(d_blockOffset, blockOffset, blocks * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Efficient::scan::hipMemcpyHostToDevice fails!");

            // Call kernAddBlockOffset
            kernAddBlockOffset<<<blocks, threads>>>(pad_n, d_odata, d_blockOffset);
            checkCUDAError("Efficient::scan::kernAddBlockOffset fails!");

            // Copy the value back
            hipMemcpy(odata, d_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Efficient::scan::hipMemcpyDeviceToHost fails!");

            //timer().endGpuTimer();

            delete[] blockSum;
            delete[] blockOffset;
            hipFree(d_idata);
            hipFree(d_odata);
            hipFree(d_blockSum);
            hipFree(d_blockOffset);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int threads = 1024;
            int blocks = (n + threads - 1) / threads; // ceil

            // Allocate memory on device
            int* d_odata, *d_idata, *d_bools, *d_indices;

            hipMalloc(&d_odata, n * sizeof(int));
            checkCUDAError("Efficient::compact::hipMalloc d_odata fails!");

            hipMalloc(&d_idata, n * sizeof(int));
            checkCUDAError("Efficient::compact::hipMalloc d_idata fails!");

            hipMalloc(&d_bools, n * sizeof(int));
            checkCUDAError("Efficient::compact::hipMalloc d_bools fails!");

            hipMalloc(&d_indices, n * sizeof(int));
            checkCUDAError("Efficient::compact::hipMalloc d_indices fails!");

            hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Efficient::compact::hipMemcpyHostToDevice fails!");

            // Create boolean array
            Common::kernMapToBoolean<<<blocks, threads>>>(n, d_bools, d_idata);
            checkCUDAError("Efficient::compact::kernMapToBoolean fails!");

            // Create indices array through exclusive scan
            scan(n, d_indices, d_bools);

            // Scatter
            Common:: kernScatter<<<blocks, threads>>>(n, d_odata, d_idata, d_bools, d_indices);
            checkCUDAError("Efficient::compact::kernScatter fails!");

            hipMemcpy(odata, d_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Efficient::compact::hipMemcpyDeviceToHost fails!");

            // Get the count
            int lastIndex, lastBool;
            hipMemcpy(&lastIndex, d_indices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastBool, d_bools + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            int count = lastIndex + lastBool;

            timer().endGpuTimer();

            hipFree(d_odata);
            hipFree(d_idata);
            hipFree(d_bools);
            hipFree(d_indices);

            return count;
        }
    }
}
